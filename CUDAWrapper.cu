
#include <hip/hip_runtime.h>
/*

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>

typedef void (*KernelFunction)(int**, int**);

// Integrate this into CUDAWrapperCall so it's more readable.
typedef VRAM_ptr void*;


//  NOTICE: This currently takes in integer values when ideally it takes in floats

// @TODO: Create an alternative method that's basically the same but with floats. Samw with the CUDA_INT_ARRAY structure


// Using this will require a re-structuring of the codebase so that prop attributes are all stored on one void* 
typedef struct CUDA_INT_ARRAY
{
    float* x;
    unsigned int rows;
    unsigned int columns;
};

cudaError_t CUDAWrapperCall(CUDA_INT_ARRAY input, CUDA_INT_ARRAY output, KernelFunction kernel)
{
    // set them all to nullptr

    int* dev_input[input.width];
    for (int i = 0; i < input.width; i++)
    {
        dev_input[i] = nullptr;
    }

    cudaError_t cudaStatus;


    // Shouldn't need to change this
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = cudaSetDevice(0);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    // Copy IO vectors to VRAM

    for (int i = 0; i < input.width; i++)
    {
        // This populates a pointer at dev_input[i] as an address in VRAM
        cudaStatus = cudaMalloc((void**)&dev_input[i], input.height * sizeof(int));
        if (cudaStatus != cudaSuccess) {
            fprintf(stderr, "cudaMalloc failed!");
            goto Error;
        }
        // Copies each array(input.x[i]) at dev_input[i]
        cudaStatus = cudaMemcpy(dev_input[i], input.x[i], input.height * sizeof(int), cudaMemcpyHostToDevice);
        if (cudaStatus != cudaSuccess) {
            fprintf(stderr, "cudaMemcpy failed!");
            goto Error;
        }
    }
    


    // Launch a kernel on the GPU with one thread for each element. dev_input is a pointer within VRAM to an array of pointers (Stored in VRAM) to individual int fields (each column)
    kernel <<<1, height >>> (dev_input, output);

    // Check for any errors launching the kernel
    cudaStatus = cudaGetLastError();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = cudaDeviceSynchronize();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = cudaMemcpy(a, dev_input[i], height * sizeof(int), cudaMemcpyDeviceToHost);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

Error:
    for (int i = 0; i < input.width; i++)
    {
        cudaFree(dev_input[i]);
    }
    


    return cudaStatus;
}
*/